

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>


struct timeval startTime, stopTime;
int started = 0;

void start_timer() {
 started = 1;
 gettimeofday(&startTime, NULL);
}

double stop_timer() {
 long seconds, useconds;
 double duration = -1;

 if (started) {
   gettimeofday(&stopTime, NULL);
   seconds  = stopTime.tv_sec  - startTime.tv_sec;
   useconds = stopTime.tv_usec - startTime.tv_usec;
   duration = (seconds * 1000.0) + (useconds / 1000.0);
   started = 0;
 }
 return duration;
}




void  printprimes(int limit, int *arr) {
  int c;
  //#pragma omp parallel for shared(arr, limit) private(c)
  for(c = 2; c <limit; c++) {
      if(arr[c] == 0) {
          fprintf(stdout,"%d ", c);
      }
  }
  fprintf(stdout,"\n");
  /* code */
}



__global__ void init(int *arr, int sqroot, int limit) {
  int c;
  for(c = 2; c <= sqroot; c++) {
      if(arr[c] == 0) {
          /*
          #pragma omp parallel for shared(arr, limit, c) private(m)
          for(m = c+1; m < limit; m++) {
              if(m%c == 0) {
                  arr[m] = 1;
              }
          }
          */
          int tid = c+1+ threadIdx.x + (blockIdx.x * blockDim.x);
          if (tid<limit){
            if (tid % c ==0) {
          		arr[tid] = 1;
          	}
          }


      }
  }
}

double getThreadAndInfo(hipDeviceProp_t devProp)
{
  /*
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %i\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    */
    return devProp.maxThreadsPerBlock;
}

int main(int argc, char **argv) {
    // Number of CUDA devices
    int threads=1000000;
    int devCount;
    hipGetDeviceCount(&devCount);
    //printf("CUDA Device Query...\n");
    //printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        //printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
       hipGetDeviceProperties(&devProp, i);
       if (getThreadAndInfo(devProp)<threads){
         threads=getThreadAndInfo(devProp);
       }

    }

    int N=10;
    int limit ;
    if (argc>3){
      fprintf(stderr, "Error: uso: %s [limite_superior_positivo]\n", argv[0]);
      return -1;

    }else if (argc==2 || argc==3) {
      int parsed=atoi(argv[1]);
      if (parsed<0){
        fprintf(stderr, "Error: uso: %s [limite_superior_positivo]\n", argv[0]);
        return -1;
      }else{
        limit=parsed;
      }
      if (argc==3) {
        N=1;
      }
    }else {
      limit=16;
    }




      int *arr;

      double ms;
      ms = 0;
      int i;
      int *p_array;
      for (i = 0; i < N; i++) {
        start_timer();
        //->
        int sqroot = (int)sqrt(limit);
        arr = (int*)malloc(limit * sizeof(int));
        hipMalloc((void**) &p_array, limit * sizeof(int));
        hipMemset(p_array, 0, limit*sizeof(int));
        if (limit<=threads){
		threads=limit;
          init<<<1, threads>>>(p_array, sqroot, limit);
        }else{
          init<<<int(limit/threads)+1, threads>>>(p_array, sqroot, limit);
        }



        hipMemcpy(arr, p_array, limit * sizeof(int), hipMemcpyDeviceToHost);

        //->
        ms += stop_timer();
      }


      if (argc==2){
        printf("times %i - avg time = %.5lf ms, %i threads\n",N,(ms / N), threads);
      }






      //




          printprimes(limit, arr);



      free(arr);

  hipFree(p_array);


    return 0;
}

